#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <vector>

#include "common.h"

#define NUM_THREADS 1024

extern double size;
//
//  benchmarking program
//

__device__ void apply_force_gpu(particle_t &particle, particle_t &neighbor)
{
  double dx = neighbor.x - particle.x;
  double dy = neighbor.y - particle.y;
  double r2 = dx * dx + dy * dy;
  if( r2 > cutoff*cutoff )
      return;
  //r2 = fmax( r2, min_r*min_r );
  r2 = (r2 > min_r*min_r) ? r2 : min_r*min_r;
  double r = sqrt( r2 );

  //
  //  very simple short-range repulsive force
  //
  double coef = ( 1 - cutoff / r ) / r2 / mass;
  particle.ax += coef * dx;
  particle.ay += coef * dy;

}

/*
__global__ void compute_forces_gpu(particle_t * particles, int n)
{
  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particles[tid].ax = particles[tid].ay = 0;
  for(int j = 0 ; j < n ; j++)
    apply_force_gpu(particles[tid], particles[j]);

}
*/

__global__ void compute_forces_gpu( particle_t &particles, int n, int &bins, int &particlesInBin, double binsize ){

    /*
        "particles" contains actual particles(structures)
        "bins" contains index of particles sorted by Bin number
        "particlesInBin" contains number of particles in each bin; helps traverse "bins"
        "n" is the number of particles in "particles"
        "binsize" is maximum binsize in 1-D (geometric-distance-wise) = cutoff
    */

    // each block computes a bin using 8 neighbors in a common case

    // get current block from global to shared memory
    __shared__ particle_t currentBin[n/2];

    // get 8 neighboring blocks from global to shared memory
    __shared__ particle_t northBin[n/2];
    __shared__ particle_t southBin[n/2];
    __shared__ particle_t eastBin[n/2];
    __shared__ particle_t westBin[n/2];
    __shared__ particle_t northEastBin[n/2];
    __shared__ particle_t northWestBin[n/2];
    __shared__ particle_t southEastBin[n/2];
    __shared__ particle_t southWestBin[n/2];

    // each thread handles a particle
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    // we are not at an edge or a corner. -- Most common case
    if( (Location.Left | Location.Right | Location.Top | Location.Bottom) == false){
        
    }
    // Top Row
    else if( Location.Top ){
        // most common case for the top row -- Not in a corner.
        if( (Location.Left | Location.Right) == false){
                    //printf("Top Row %d \n", BinIndex );
 
        }
        else if( (!Location.Left) && Location.Right ){
                    //printf("Top Row Right %d \n", BinIndex );
                    // Right == East

        }
        else if ( Location.Left && (!Location.Right) ){
                    //printf("Top Row Left %d \n", BinIndex );

        }

    }
    else if( Location.Bottom ){
        // most common case for the top row -- Not in a corner.
        if((Location.Left | Location.Right) == false){
                    //printf("Bottom Row %d \n", BinIndex );

        }
        else if( (!Location.Left) && Location.Right ){
                    // Right == East
                    //printf("Bottom Row Right %d \n", BinIndex );

        }
        else if( Location.Left && (!Location.Right) ){
                    //printf("Bottom Row Left %d ", BinIndex );

        }

    }
    else if(Location.Left){
                //printf("Left %d \n", BinIndex );

    }
    else if(Location.Right){
                //printf("Right %d \n", BinIndex );

    }
    else{
        printf("Getting another bin case\n");
    }

}

__global__ void move_gpu (particle_t * particles, int n, double size)
{

  // Get thread (particle) ID
  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  if(tid >= n) return;

  particle_t * p = &particles[tid];
    //
    //  slightly simplified Velocity Verlet integration
    //  conserves energy better than explicit Euler method
    //
    p->vx += p->ax * dt;
    p->vy += p->ay * dt;
    p->x  += p->vx * dt;
    p->y  += p->vy * dt;

    //
    //  bounce from walls
    //
    while( p->x < 0 || p->x > size )
    {
        p->x  = p->x < 0 ? -(p->x) : 2*size-p->x;
        p->vx = -(p->vx);
    }
    while( p->y < 0 || p->y > size )
    {
        p->y  = p->y < 0 ? -(p->y) : 2*size-p->y;
        p->vy = -(p->vy);
    }

}



int main( int argc, char **argv )
{    
    // This takes a few seconds to initialize the runtime
    hipDeviceSynchronize(); 

    if( find_option( argc, argv, "-h" ) >= 0 )
    {
        printf( "Options:\n" );
        printf( "-h to see this help\n" );
        printf( "-n <int> to set the number of particles\n" );
        printf( "-o <filename> to specify the output file name\n" );
        return 0;
    }
    
    int n = read_int( argc, argv, "-n", 1000 );

    char *savename = read_string( argc, argv, "-o", NULL );
    
    FILE *fsave = savename ? fopen( savename, "w" ) : NULL;
    particle_t *particles = (particle_t*) malloc( n * sizeof(particle_t) );

    // GPU particle data structure
    particle_t * d_particles;
    hipMalloc((void **) &d_particles, n * sizeof(particle_t));

    set_size( n );
    init_particles( n, particles );

    double size = getSize(); 
    int NumofBinsEachSide = getNumberofBins(size);
    int NumofBins = NumofBinsEachSide*NumofBinsEachSide;
    
    double binsize = getBinSize();

    std::vector< std::vector<int> > Bins(NumofBins, std::vector<int>(0));

    hipDeviceSynchronize();
    double copy_time = read_timer( );

    // Copy the particles to the GPU
    hipMemcpy(d_particles, particles, n * sizeof(particle_t), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
    copy_time = read_timer( ) - copy_time;

    
    //
    //  simulate a number of time steps
    //
    hipDeviceSynchronize();
    double simulation_time = read_timer( );

    for( int step = 0; step < NSTEPS; step++ )
    {
        // clear bins for the iteration
        for(int clear = 0; clear < Bins.size(); clear++ ){
            Bins[clear].clear();
        }

        for(int particleIndex = 0; particleIndex < n; ++particleIndex){

            // place particles in bins
            int BinX = (int)(particles[particleIndex].x/binsize);
            int BinY = (int)(particles[particleIndex].y/binsize);

            int BinNum = BinX + NumofBinsEachSide*BinY;
            
            Bins[BinNum].push_back(particleIndex);
        }

        /// Beginning conversion from STL to primitives ///

        // Find number of particles in each bin
        int* particlesInBin = (int*) malloc( sizeof(int) * NumofBins );
        int prev_p=0;
        for( int b = 0; b < NumofBins; ++b){
            particlesInBin[b] = Bins[b].size();
            //totalSize += particlesInBin[b];

            int p;
            // prepare bins array from 2d Bins vector
            for( p=0; p < particlesInBin[b]; ++p ){
                bins[prev_p + p] = Bins[b].at(p);
            }
            prev_p += p;
        }
        int totalSize = prev_p;

        void* d_pib;
        hipMalloc( &d_pib, sizeof(int) * NumofBins );
        int* d_particlesInBin = d_pib;

        hipMemcpy( d_particlesInBin, particlesInBin, sizeof(int) * NumofBins, hipMemcpyHostToDevice );

        void* B;
        hipMalloc( &B, sizeof(int) * totalSize );
        int* d_bins = B;

        hipMemcpy( d_bins, bins, sizeof(int) * totalSize, hipMemcpyHostToDevice );

        //int blks = (n + NUM_THREADS - 1) / NUM_THREADS;
	compute_forces_gpu <<< NumofBins, NUM_THREADS >>> (d_particles, n, d_bins, d_particlesInBin, binsize);
        
        //
        //  move particles
        //
	move_gpu <<< blks, NUM_THREADS >>> (d_particles, n, size);
        
        //
        //  save if necessary
        //
        if( fsave && (step%SAVEFREQ) == 0 ) {
	    // Copy the particles back to the CPU
            hipMemcpy(particles, d_particles, n * sizeof(particle_t), hipMemcpyDeviceToHost);
            save( fsave, n, particles);
	}
    }
    hipDeviceSynchronize();
    simulation_time = read_timer( ) - simulation_time;
    
    printf( "CPU-GPU copy time = %g seconds\n", copy_time);
    printf( "n = %d, simulation time = %g seconds\n", n, simulation_time );
    
    free( particles );
    hipFree(d_particles);
    if( fsave )
        fclose( fsave );
    
    return 0;
}
